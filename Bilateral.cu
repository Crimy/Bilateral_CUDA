#include "hip/hip_runtime.h"
#include "Bilateral.h"

__global__ void BGRToGrayscale(const uchar* Input, int* Output);
__global__ void BilateralFilter(const int* Input, uchar* Output);

Bilateral_Filter::Bilateral_Filter() {
	hipMemcpyToSymbolAsync(HIP_SYMBOL(PI_), &PI, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(BILATERAL_), &BILATERAL, sizeof(Bilateral), 0, hipMemcpyHostToDevice);
}

Bilateral_Filter::~Bilateral_Filter() {
	Image_Bilateral.~Mat();

	hipFree(Image_CUDA);
	hipFree(Gray_CUDA);
}

void Bilateral_Filter::InitSetting(const cv::Mat Image) {
	const int x = Image.cols;
	const int y = Image.rows;

	VIDEO_XY[0] = x;
	VIDEO_XY[1] = y;
	VIDEO_SIZE = x * y;

	BlocksPerGrid = dim3((int)ceilf((float)VIDEO_XY[0] / (float)ThreadsPerBlock.x),
						(int)ceilf((float)VIDEO_XY[1] / (float)ThreadsPerBlock.y));

	hipMemcpyToSymbolAsync(HIP_SYMBOL(VIDEO_XY_), VIDEO_XY, 2 * sizeof(int), 0, hipMemcpyHostToDevice);
	hipMalloc((void**)&Image_CUDA, VIDEO_SIZE * 3 * sizeof(uchar));
	hipMalloc((void**)&Gray_CUDA, VIDEO_SIZE * sizeof(int));

	Image_Bilateral = cv::Mat(cv::Size(x, y), CV_8UC1);
}

void Bilateral_Filter::Operate(const cv::Mat Image) {
	hipMemcpyAsync(Image_CUDA, Image.data, VIDEO_SIZE * 3 * sizeof(uchar), hipMemcpyHostToDevice);

	BGRToGrayscale_(Image_CUDA, Gray_CUDA);
	BilateralFilter_(Gray_CUDA, Image_Bilateral.data);

	hipDeviceSynchronize();

	cv::imshow("Result", Image_Bilateral);
}

void Bilateral_Filter::BGRToGrayscale_(const uchar* CUDA_In, int* CUDA_Out) {
	BGRToGrayscale << <BlocksPerGrid, ThreadsPerBlock >> > (CUDA_In, CUDA_Out);
}

void Bilateral_Filter::BilateralFilter_(const int* CUDA_In, uchar* CUDA_Out) {
	BilateralFilter << <BlocksPerGrid, ThreadsPerBlock >> > (CUDA_In, CUDA_Out);

}

__global__ void BGRToGrayscale(const uchar* Input, int* Output) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x >= VIDEO_XY_[0] || y >= VIDEO_XY_[1])
		return;

	int idx = y * VIDEO_XY_[0] + x;

	Output[idx] = BGRToGrayscale_Device(Input, idx);
}

__global__ void BilateralFilter(const int* Input, uchar* Output) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x >= VIDEO_XY_[0] || y >= VIDEO_XY_[1])
		return;

	int idx = y * VIDEO_XY_[0] + x;

	__shared__ uchar Pixel;

	if (x < 2 || x > VIDEO_XY_[0] - 3 || y < 2 || y > VIDEO_XY_[1] - 3) {
		Pixel = (uchar)Input[idx];
		Output[idx] = Pixel;
		return;
	}
	
	Pixel = (uchar)BilateralFilter_Device(Input, idx);
	Output[idx] = Pixel;
}